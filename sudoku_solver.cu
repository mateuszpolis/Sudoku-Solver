#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>

#define SIZE 9
#define NUM_THREADS 256
#define MAX_BOARDS 10000000 // Adjust as needed
#define MAX_ITERATIONS 1000 // Safety limit to prevent infinite loops

// Device function to check if placing num at (row, col) is valid
__device__ bool is_valid(int* board, int row, int col, int num) {
    // Check row
    for (int i = 0; i < SIZE; ++i)
        if (board[row * SIZE + i] == num)
            return false;

    // Check column
    for (int i = 0; i < SIZE; ++i)
        if (board[i * SIZE + col] == num)
            return false;

    // Check 3x3 grid
    int startRow = (row / 3) * 3;
    int startCol = (col / 3) * 3;
    for (int i = 0; i < 3; ++i)
        for (int j = 0; j < 3; ++j)
            if (board[(startRow + i) * SIZE + startCol + j] == num)
                return false;

    return true;
}

// Modify the kernel as follows:

__global__ void solve_sudoku_kernel(
    int* current_boards, 
    int* next_boards, 
    int* next_count, 
    int* solutions, 
    int* solution_count, 
    int num_current_boards
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_current_boards) return;

    int board[SIZE * SIZE];
    for (int i = 0; i < SIZE * SIZE; ++i)
        board[i] = current_boards[idx * SIZE * SIZE + i];

    // Find the first empty cell
    int empty_pos = -1;
    for (int i = 0; i < SIZE * SIZE; ++i) {
        if (board[i] == 0) {
            empty_pos = i;
            break;
        }
    }

    // If no empty cell, record solution
    if (empty_pos == -1) {
        int sol_idx = atomicAdd(solution_count, 1);
        if (sol_idx < MAX_BOARDS) { // Check we don't exceed our solution buffer
            for (int i = 0; i < SIZE * SIZE; ++i)
                solutions[sol_idx * SIZE * SIZE + i] = board[i];
        } else {
            // If we exceed the solution buffer, we might stop or just skip writing
            // Consider reverting the atomicAdd if desired, but that's tricky
            // For simplicity, we will just skip writing
        }
        return;
    }

    int row = empty_pos / SIZE;
    int col = empty_pos % SIZE;
    bool any_valid = false;

    for (int num = 1; num <= SIZE; ++num) {
        if (is_valid(board, row, col, num)) {
            any_valid = true;
            int new_board_idx = atomicAdd(next_count, 1);
            if (new_board_idx < MAX_BOARDS) {
                // Write the new board
                for (int i = 0; i < SIZE * SIZE; ++i)
                    next_boards[new_board_idx * SIZE * SIZE + i] = board[i];
                next_boards[new_board_idx * SIZE * SIZE + empty_pos] = num;
            } else {
                // If we exceed MAX_BOARDS for expansions, we should consider stopping further expansions
                // One strategy is to just skip creating more boards
                // Another is to reset next_count back by one atomic operation, but this can be complex
                // For now, we just skip writing more boards
            }
        }
    }
}

int main() {
    // Read boards from a file
    std::string filename = "boards.txt";
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        std::cerr << "Failed to open " << filename << " for reading.\n";
        return 1;
    }

    int num_boards;
    infile >> num_boards;
    if (num_boards <= 0) {
        std::cerr << "No boards to solve.\n";
        return 1;
    }

    // Read all boards into a host vector
    // We'll store all boards consecutively: board0(81 ints), board1(81 ints), ...
    std::vector<int> host_boards(num_boards * SIZE * SIZE);
    for (int b = 0; b < num_boards; ++b) {
        for (int i = 0; i < SIZE; ++i) {
            std::string line;
            infile >> line;
            if (line.size() != SIZE) {
                std::cerr << "Invalid board line encountered.\n";
                return 1;
            }
            for (int j = 0; j < SIZE; ++j) {
                char c = line[j];
                if (c < '0' || c > '9') {
                    std::cerr << "Invalid character in board: " << c << std::endl;
                    return 1;
                }
                int val = c - '0';
                host_boards[b * SIZE * SIZE + i * SIZE + j] = val;
            }
        }
    }

    infile.close();

    // Allocate memory for current boards and next boards on the GPU
    int* d_current_boards;
    int* d_next_boards;
    hipMalloc(&d_current_boards, sizeof(int) * SIZE * SIZE * MAX_BOARDS);
    hipMalloc(&d_next_boards, sizeof(int) * SIZE * SIZE * MAX_BOARDS);

    // Allocate memory for solutions on the GPU
    int* d_solutions;
    hipMalloc(&d_solutions, sizeof(int) * SIZE * SIZE * MAX_BOARDS); // Adjust size as needed

    // Allocate memory for counts
    int* d_next_count;
    int* d_solution_count;
    hipMalloc(&d_next_count, sizeof(int));
    hipMalloc(&d_solution_count, sizeof(int));

    // Initialize next_count and solution_count to zero
    hipMemset(d_next_count, 0, sizeof(int));
    hipMemset(d_solution_count, 0, sizeof(int));

    // Copy all boards to current_boards on device
    hipMemcpy(d_current_boards, host_boards.data(), sizeof(int) * SIZE * SIZE * num_boards, hipMemcpyHostToDevice);

    // Initialize the number of current boards to the number of boards we read
    int num_current_boards = num_boards;

    // Define kernel launch parameters
    int threads_per_block = NUM_THREADS;

    int iteration = 0; // Iteration counter

    auto start_algorithm = std::chrono::high_resolution_clock::now();

    // Iterate until all boards are processed or maximum iterations reached
    while (num_current_boards > 0 && iteration < MAX_ITERATIONS) {
        iteration++;

        // Reset next_count to zero
        hipMemset(d_next_count, 0, sizeof(int));

        // Launch kernel to process current boards
        solve_sudoku_kernel<<<(num_current_boards + threads_per_block - 1) / threads_per_block, threads_per_block>>>(
            d_current_boards,
            d_next_boards,
            d_next_count,
            d_solutions,
            d_solution_count,
            num_current_boards
        );

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed at iteration " << iteration << ": " 
                      << hipGetErrorString(err) << std::endl;
            break;
        }

        // Synchronize to ensure kernel completion
        hipDeviceSynchronize();

        // Get the number of next boards
        int h_next_count;
        hipMemcpy(&h_next_count, d_next_count, sizeof(int), hipMemcpyDeviceToHost);

        // Get the number of solutions found so far
        int h_solution_count;
        hipMemcpy(&h_solution_count, d_solution_count, sizeof(int), hipMemcpyDeviceToHost);

        if (h_next_count >= MAX_BOARDS) {
            std::cerr << "MAX_BOARDS limit reached. Stopping expansions." << std::endl;            
            break;
        }

        // Swap current_boards and next_boards
        int* temp = d_current_boards;
        d_current_boards = d_next_boards;
        d_next_boards = temp;

        // Update the number of current boards for the next iteration
        num_current_boards = h_next_count;

        // If no new boards are generated, exit the loop
        if (h_next_count == 0) {
            break;
        }
    }

    auto end_algorithm = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration_algorithm = end_algorithm - start_algorithm;
    // std::cout << "Time taken for important algorithm stages: " << duration_algorithm.count() << " seconds\n";
    std::cout << duration_algorithm.count() << "\n";

    // Get the total number of solutions
    int total_solutions;
    hipMemcpy(&total_solutions, d_solution_count, sizeof(int), hipMemcpyDeviceToHost);

    // Allocate host memory for solutions
    std::vector<int> solutions(total_solutions * SIZE * SIZE);
    if (total_solutions > 0) {
        hipMemcpy(solutions.data(), d_solutions, sizeof(int) * SIZE * SIZE * total_solutions, hipMemcpyDeviceToHost);
    }

    // Open the output file
    std::ofstream outfile("solutions.txt");
    if (!outfile.is_open()) {
        std::cerr << "Failed to open solutions.txt for writing.\n";
        // Free memory before exiting
        hipFree(d_current_boards);
        hipFree(d_next_boards);
        hipFree(d_solutions);
        hipFree(d_next_count);
        hipFree(d_solution_count);
        return 1;
    }

    // Write the total number of solutions
    outfile << "Total Solutions Found: " << total_solutions << "\n\n";

    // Write each solution to the file
    for (int s = 0; s < total_solutions; ++s) {
        outfile << "Solution " << s + 1 << ":\n";
        for (int i = 0; i < SIZE; ++i) {
            for (int j = 0; j < SIZE; ++j) {
                outfile << solutions[s * SIZE * SIZE + i * SIZE + j] << " ";
            }
            outfile << "\n";
        }
        outfile << "\n";
    }

    // Close the file
    outfile.close();
    // std::cout << "Solutions have been saved to solutions.txt\n";

    // Free memory
    hipFree(d_current_boards);
    hipFree(d_next_boards);
    hipFree(d_solutions);
    hipFree(d_next_count);
    hipFree(d_solution_count);

    // Check if maximum iterations were reached
    if (iteration >= MAX_ITERATIONS) {
        std::cerr << "Reached maximum number of iterations (" << MAX_ITERATIONS << "). "
                  << "The program may be stuck in an infinite loop.\n";
    }

    return 0;
}